#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int arg, char **argv) {
    int nElem = 1024;
    dim3 block(1024);
    dim3 grid((nElem - 1) / block.x + 1);
    printf("grid.x %d block.x %d\n", grid.x, block.x);

    block.x = 512;
    grid.x = (nElem - block.x) / block.x;
    printf("grid.x %d block.x %d\n", grid.x, block.x);


    hipDeviceReset();

    return 0;

}