
#include <hip/hip_runtime.h>
#include "iostream"

__global__ void sayHelloWorld();

int main(){

    printf("HelloWorld! CPU \n");
    sayHelloWorld<<<2, 1>>>();

    hipDeviceReset();

    system("pause");
    return 0;
}

__global__ void sayHelloWorld(){
    printf("HelloWorld! GPU \n");
}