
#include <hip/hip_runtime.h>
#include "iostream"

__global__ void sayHelloWorld();

int main(){
    printf("HelloWorld! CPU \n");
    sayHelloWorld<<<2, 1>>>();

    hipDeviceReset();

    return 0;
}

__global__ void sayHelloWorld(){
    printf("HelloWorld! GPU \n");
}